#include "hip/hip_runtime.h"
#include "sha3_gpu.h"
#include "hip/hip_runtime_api.h"
#include "common.h"
#include <cstdlib>
#include <cassert>
#include <limits>
#include <stdexcept>

namespace
{

constexpr size_t npos = std::numeric_limits<size_t>::max();

// Array of indicies and rotation values for P and Pi phases.
__constant__ uint8_t g_ppi_aux[25][2];

// Array of indices for ksi phase.
__constant__ uint8_t g_ksi_aux[25][2];

__constant__ uint64_t g_iota_aux[24];

bool inited = false;

void init_constants()
{
  const std::array<std::pair<uint8_t, uint8_t>, 25> h_ppi_aux = {
      {{0, 0},   {6, 44},  {12, 43}, {18, 21}, {24, 14}, {3, 28},  {9, 20}, {10, 3}, {16, 45},
       {22, 61}, {1, 1},   {7, 6},   {13, 25}, {19, 8},  {20, 18}, {4, 27}, {5, 36}, {11, 10},
       {17, 15}, {23, 56}, {2, 62},  {8, 55},  {14, 39}, {15, 41}, {21, 2}}};
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_ppi_aux), h_ppi_aux.data(), h_ppi_aux.size() * sizeof(uint8_t) * 2));

  const std::array<std::pair<uint8_t, uint8_t>, 25> h_ksi_aux = {
      {{1, 2},   {2, 3},   {3, 4},   {4, 0},   {0, 1},   {6, 7},   {7, 8},   {8, 9},   {9, 5},
       {5, 6},   {11, 12}, {12, 13}, {13, 14}, {14, 10}, {10, 11}, {16, 17}, {17, 18}, {18, 19},
       {19, 15}, {15, 16}, {21, 22}, {22, 23}, {23, 24}, {24, 20}, {20, 21}}};
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_ksi_aux), h_ksi_aux.data(), h_ksi_aux.size() * sizeof(uint8_t) * 2));

  const std::array<uint64_t, 24> h_iota_aux = {
      0x0000000000000001L, 0x0000000000008082L, 0x800000000000808aL, 0x8000000080008000L, 0x000000000000808bL,
      0x0000000080000001L, 0x8000000080008081L, 0x8000000000008009L, 0x000000000000008aL, 0x0000000000000088L,
      0x0000000080008009L, 0x000000008000000aL, 0x000000008000808bL, 0x800000000000008bL, 0x8000000000008089L,
      0x8000000000008003L, 0x8000000000008002L, 0x8000000000000080L, 0x000000000000800aL, 0x800000008000000aL,
      0x8000000080008081L, 0x8000000000008080L, 0x0000000080000001L, 0x8000000080008008L};

  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_iota_aux), h_iota_aux.data(), h_iota_aux.size() * sizeof(uint64_t)));
}


__device__ uint64_t rotate(uint64_t val, unsigned n) { return val << n | val >> (64 - n); }

__device__ void processState(uint64_t *A)
{
  const size_t t = threadIdx.x;
  const size_t s = threadIdx.x % 5;

  __shared__ uint64_t C[25];

  assert(t < 25);

#pragma unroll
  for (int round_idx = 0; round_idx < 24; ++round_idx)
  {
    // Thetta phase.
    C[t] = A[s] ^ A[s + 5] ^ A[s + 10] ^ A[s + 15] ^ A[s + 20];
    A[t] ^= C[s + 5 - 1] ^ rotate(C[s + 1], 1);

    // P and Pi combined phases.
    C[t] = rotate(A[g_ppi_aux[t][0]], g_ppi_aux[t][1]);

    // Ksi phase.
    A[t] = C[t] ^ (~C[g_ksi_aux[t][0]] & C[g_ksi_aux[t][1]]);

    // Iota phase.
    A[t] ^= t == 0 ? g_iota_aux[round_idx] : 0;
  }
}

__global__ void processBlockDevice(const uint64_t *data, size_t singleBufSize, const uint64_t *end,
                                   uint64_t *A_original)
{
  const size_t t = threadIdx.x;

  __shared__ uint64_t A[25];

  if (t < 25)
  {
    A[t] = A_original[t];
    for (; data != end; data += singleBufSize)
    {
      if (t < singleBufSize)
      {
        // Apply data to inner state. Nvidia keeps all data in little-endian.
        A[t] ^= data[t];
      }
      processState(A);
    }
    A_original[t] = A[t];
  }
}

void addPadding(uint8_t *d_begin, uint8_t *d_end)
{
  const int maxBuf = 144;
  assert(d_end > d_begin);
  size_t size = d_end - d_begin;
  assert(size <= maxBuf);

  uint8_t buf[maxBuf] = {};
  if (size == 1)
  {
    buf[0] = 0x86;
  }
  else
  {
    buf[0] = 0x06;
    buf[size - 1] = 0x80;
  }

  checkCudaErrors(hipMemcpy(d_begin, buf, size, hipMemcpyHostToDevice));
}

} // namespace


SHA3_gpu::~SHA3_gpu()
{
  checkCudaErrors(hipFree(m_d_blockBuffers));
  checkCudaErrors(hipFree(m_d_A));
}

SHA3_gpu::SHA3_gpu(size_t size)
  : m_digestSize(size / 8)
{
  assert(m_digestSize * 8 == size);
  if (!inited)
  {
    init_constants();
  }
  checkCudaErrors(hipMalloc(&m_d_A, 25 * 8));
  m_singleBufSz = 200 - 2 * m_digestSize;

  checkCudaErrors(hipMalloc(&m_d_blockBuffers, m_singleBufSz * m_nBuffers));
  checkCudaErrors(hipMemset(m_d_blockBuffers, 0, m_singleBufSz * m_nBuffers));

  init();
}

void SHA3_gpu::init()
{
  checkCudaErrors(hipMemset(m_d_A, 0, 25 * sizeof(uint64_t)));
  m_bufferOffset = 0;

  m_finished = false;
}

void SHA3_gpu::add(const uint8_t *data, size_t sz)
{
  assert(!m_finished && "Init should be called");
  size_t blockSz = m_nBuffers * m_singleBufSz;
  while (sz != 0)
  {
    if (sz < blockSz - m_bufferOffset)
    {
      checkCudaErrors(hipMemcpy(m_d_blockBuffers + m_bufferOffset, data, sz, hipMemcpyHostToDevice));
      m_bufferOffset += sz;
      return;
    }

    size_t dataSize = blockSz - m_bufferOffset;
    checkCudaErrors(hipMemcpy(m_d_blockBuffers + m_bufferOffset, data, dataSize, hipMemcpyHostToDevice));
    processBlock(blockSz);
    m_bufferOffset = 0;
    sz -= dataSize;
    data += dataSize;
  }
}

std::vector<uint8_t> SHA3_gpu::digest()
{
  if (!m_finished)
  {
    size_t size = (m_bufferOffset / m_singleBufSz + 1) * m_singleBufSz;
    addPadding(m_d_blockBuffers + m_bufferOffset, m_d_blockBuffers + size);
    processBlock(size);
    m_finished = true;
  }

  std::vector<uint8_t> result(m_digestSize);
  checkCudaErrors(hipMemcpy(result.data(), m_d_A, m_digestSize, hipMemcpyDeviceToHost));
  return result;
}

void SHA3_gpu::processBlock(size_t bufSize)
{
  assert(bufSize % m_singleBufSz == 0);
  auto ptr64 = reinterpret_cast<const uint64_t *>(m_d_blockBuffers);
  assert(m_singleBufSz % 8 == 0);
  processBlockDevice<<<1, 32>>>(ptr64, m_singleBufSz / 8, ptr64 + bufSize / 8, m_d_A);
}

//
// SHA3_gpu_batch
//

struct SHA3_gpu_batch::State
{
  uint64_t *d_A;
  uint8_t *d_blockBuffer;
  size_t bufferSize = 0; // Buffer's payload size.
};

namespace
{

__global__ void processBatchBlockDevice(SHA3_gpu_batch::State *data, size_t blockSize)
{
  int t = threadIdx.x;
  int b = blockIdx.x;
  __shared__ uint64_t A[25];

  if (t < 25)
  {
    size_t bufSize = data[b].bufferSize / 8;
    A[t] = data[b].d_A[t];
    const uint64_t *buf = reinterpret_cast<const uint64_t *>(data[b].d_blockBuffer);

    for (; bufSize != 0; bufSize -= blockSize, buf += blockSize)
    {
      if (t < blockSize)
      {
        A[t] ^= buf[t];
      }
      processState(A);
    }
    data[b].d_A[t] = A[t];
  }
}

} // namespace

SHA3_gpu_batch::SHA3_gpu_batch(size_t block)
  : m_digestSize(block / 8)
  , m_singleBlockSize(200 - 2 * m_digestSize)
{
  assert(m_digestSize * 8 == block);
  if (!inited)
  {
    init_constants();
  }

  int device;
  checkCudaErrors(hipGetDevice(&device));

  hipDeviceProp_t props;
  checkCudaErrors(hipGetDeviceProperties(&props, device));
  int cores = props.major == 9999 && props.minor == 9999 ? 1 : _ConvertSMVer2Cores(props.major, props.minor);
  cores *= props.multiProcessorCount;
  m_nBlocks = cores / props.warpSize;
  m_states = std::make_unique<State[]>(m_nBlocks);
  checkCudaErrors(hipMalloc(&m_d_states, m_nBlocks * sizeof(State)));
  size_t aSize = 25 * sizeof(uint64_t);
  size_t available;
  checkCudaErrors(hipMemGetInfo(&available, nullptr));

  size_t nSingleBuffers = (available - aSize * m_nBlocks) / m_nBlocks / m_singleBlockSize;
  if (nSingleBuffers == 0)
  {
    throw std::logic_error("Not enough memory on gpu device. Please, select another one");
  }

  // std::min takes reference and m_maxBuffers is not placed.
  // Create new value, that equals to m_maxBuffers.
  nSingleBuffers = std::min(nSingleBuffers, size_t(m_maxBuffers));

  m_bufferSize = nSingleBuffers * m_singleBlockSize;

  for (size_t i = 0; i < m_nBlocks; ++i)
  {
    checkCudaErrors(hipMalloc(&m_states[i].d_A, aSize));
    checkCudaErrors(hipMalloc(&m_states[i].d_blockBuffer, m_bufferSize));
    assert(m_states[i].d_blockBuffer != nullptr);
  }
}

SHA3_gpu_batch::~SHA3_gpu_batch()
{
  for (size_t i = 0; i < m_nBlocks; ++i)
  {
    checkCudaErrors(hipFree(m_states[i].d_A));
    checkCudaErrors(hipFree(m_states[i].d_blockBuffer));
  }
  checkCudaErrors(hipFree(m_d_states));
}

std::vector<SHA3_gpu_batch::Digest>
    SHA3_gpu_batch::calculate(const std::vector<std::pair<const uint8_t *, size_t>> &datas)
{
  struct LocalState
  {
    size_t idx = npos; // index of processed element
    size_t globalOffset = 0;
  };

  std::vector<SHA3_gpu_batch::Digest> result = prepareResult(datas.size());

  size_t loopSize = std::min<size_t>(m_nBlocks, datas.size());

  std::vector<LocalState> localState(loopSize);

  size_t next = 0;
  size_t finished = 0;
  while (finished < datas.size())
  {
    for (size_t i = 0; i < loopSize; ++i)
    {
      // Task distributing.
      if (localState[i].idx == npos)
      {
        if (next >= datas.size())
        {
          // Nothing to give.
          continue;
        }
        localState[i].idx = next++;
        localState[i].globalOffset = 0;
        checkCudaErrors(hipMemset(m_states[i].d_A, 0, 25 * 8));
      }

      // Fill buffers.
      auto &from = datas[localState[i].idx];
      size_t nCopy = std::min(from.second - localState[i].globalOffset, m_bufferSize);

      checkCudaErrors(hipMemcpy(m_states[i].d_blockBuffer, from.first + localState[i].globalOffset, nCopy,
                                 hipMemcpyHostToDevice));

      if (nCopy != m_bufferSize)
      {
        // We didn't fill the whole buffer => it's data end. We need to add padding to the last block.
        size_t newOffset = (1 + nCopy / m_singleBlockSize) * m_singleBlockSize;
        assert(newOffset <= m_bufferSize);
        addPadding(m_states[i].d_blockBuffer + nCopy, m_states[i].d_blockBuffer + newOffset);

        nCopy = newOffset;
      }

      m_states[i].bufferSize = nCopy;
      localState[i].globalOffset += nCopy;
    }

    launchKernel();

    for (size_t i = 0; i < loopSize; ++i)
    {
      if (localState[i].idx != npos && localState[i].globalOffset > datas[localState[i].idx].second)
      {
        // Collect results
        checkCudaErrors(
            hipMemcpy(result[localState[i].idx].data(), m_states[i].d_A, m_digestSize, hipMemcpyDeviceToHost));
        localState[i].idx = npos;
        // Mark state as empty for cases when there is no work to do.
        // This 0 is required for gpu not to perform inappropriate work.
        m_states[i].bufferSize = 0;
        ++finished;
      }
    }
  }
  return result;
}

std::vector<SHA3_gpu_batch::Digest> SHA3_gpu_batch::prepareResult(size_t size)
{
  std::vector<Digest> result;
  result.resize(size);
  for (size_t i = 0; i < size; ++i)
  {
    result[i].resize(m_digestSize);
  }
  return result;
}

void SHA3_gpu_batch::launchKernel()
{
  if (!isLittleEndian())
  {
    for (size_t i = 0; i < m_nBlocks; ++i)
    {
      m_states[i].bufferSize = toLittleEndian(m_states[i].bufferSize);
    }
  }

  checkCudaErrors(hipMemcpy(m_d_states, m_states.get(), m_nBlocks * sizeof(State), hipMemcpyHostToDevice));
  processBatchBlockDevice<<<m_nBlocks, 32>>>(m_d_states, m_singleBlockSize / 8);
#ifndef NDEBUG
  checkCudaErrors(hipDeviceSynchronize());
#endif
}
